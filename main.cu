
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#define leitura 1
/*
__global__ void algoritmo(int* mat, int* res, int size, int* total_feromonios, int tamanho_menor_resposta){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int* marca = (int*) calloc(1,size * sizeof(int));
    int posicoes = size;

    int pos = rand() % size;
    int fer_ini = (rand() % total_feromonios[pos]) + 1;
    pos = -1;

    while(fer_ini > 0){
        ++pos;
        fer_ini -= mat[tid*size + pos];
    }

    marca[pos] = 1;
    --posicoes;

    for(int i = 0; i < size; ++i){
        if(mat[pos*size + i] == 0){
            marca[i] = 1;
            --posicoes;
        }
    }

    int controle;
    while(posicoes > 0){
        controle = 0;
        fer_ini = (rand() % total_feromonios[pos]) + 1;
        for(int i = 0; i < size; ++i){
            controle += mat[tid*size]
        }
    }

}
*/

__global__ void setup_kernel(hiprandState *state, unsigned long seed) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, id, 0, &state[id]);
}

__global__ void algoritmo(int* mat, int* res, int size, int* feromonios, int* menorGlobal, int* posicaoMenorGlobal, hiprandState *state, int* validacao, int* g_resFinal){
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int val = tid * size;
    if(tid < size){
        // Se aux == 0 a função deve fechar
        int aux = size;

        int menorLocal = 0;

        // Criando o vetor de validacao
        //int* validacao = (int*) malloc(size * sizeof(int));

        for(int i = 0; i < size; ++i){
            validacao[val + i] = 0;
            res[val + i] = 0;
        }

        // Seleciona aleatoriamente o primeiro vertice
        int pontape = hiprand(&state[tid]) % size;

        // Marcando que ja passou pelo pontape
        validacao[val + pontape] = 1;
        res[tid*size] = pontape;
        ++menorLocal;
        --aux;

        do{
            // Remover todos os valores invalidos
            for(int i = 0; i < size; ++i){
                if(mat[pontape*size + i] == 0 && validacao[val + i] != 1){
                    validacao[val + i] = 1;
                    --aux;
                }
            }

            // Seleciona aleatoriamente a primeira aresta
            int valor_caminhar_feromonios = hiprand(&state[tid]) % feromonios[pontape];

            int contador = 0;
            int valor_temporario = -1;
            for(int i = 0; i < size; ++i){
                contador += mat[pontape*size + i];
                if(validacao[val + i] == 0){
                    if(contador >= valor_caminhar_feromonios){
                        pontape = i;
                        validacao[val + pontape] = 1;
                        res[tid*size + menorLocal] = pontape;
                        ++menorLocal;
                        --aux;    
                        break;            
                    }else{
                        valor_temporario = i;
                    }
                }
                if(contador >= valor_caminhar_feromonios && valor_temporario != -1){
                    pontape = valor_temporario;
                    validacao[val + pontape] = 1;
                    res[tid*size + menorLocal] = pontape;
                    ++menorLocal;
                    --aux;
                    break;
                }
            }
        }while(aux > 0);

        res[tid*size + menorLocal] = -1;        

        if(menorLocal < menorGlobal[0]){
            menorGlobal[0] = menorLocal;
            posicaoMenorGlobal[0] = tid;
            int i = 0;
            while(res[tid*size + i] != -1){
                g_resFinal[i] = res[tid*size + i];
                ++i;
            }
            g_resFinal[i] = -1;
            //res[tid*size + menorLocal] = -1;
        }

    }
    //__syncthreads();
}

__global__ void atualizaFeromonios(int* mat, int* res, int* posMenor, int size, int* feromonios){
    int pivo = 0;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(tid < size){
        while(res[pivo + 1] != -1){
            mat[res[pivo]*size + res[pivo + 1]] += 20;
            mat[res[pivo + 1]*size + res[pivo]] += 20;
            feromonios[res[pivo]] += 20;
            feromonios[res[pivo + 1]] += 20;
            ++pivo;
        }        
    }

}

int main(){


    FILE *file;
    file = fopen("instancia_teste.mis", "r");
    int size, vertices;

    srand(time(NULL));
    clock_t start, end;
    double cpu_time_used;

    if (leitura == 1)
    {
        fscanf(file, "%i %i\n", &size, &size);
        fscanf(file, "1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 1 ");
    }
    else if (leitura == 2)
    {
        fscanf(file, "p edge %i %i    ", &size, &vertices);
    }

    int* c_mat = (int*) malloc(size * size * sizeof(int));
    int* c_res = (int*) malloc(size * size * sizeof(int));
    int* feromoniosVetor = (int*) calloc(1, size * sizeof(int));
    for(int i = 0; i < size; ++i){
        //total_feromonios = 0;
        for(int j = 0; j < size; ++j){
            c_mat[(i*size) + j] = 1;
            //++total_feromonios;
        }
        //feromoniosVetor[i] = total_feromonios;
    }

    int valor1, valor2;

    if (leitura == 1){
        while (!feof(file))
        {
            fscanf(file, "%i %i\n", &valor1, &valor2);
            // printf("%i %i\n", valor1, valor2);
            //  fscanf(file,"%i %i\n", &valor1, &valor2);
            --valor1;
            --valor2;
            c_mat[valor1*size + valor2] = 0;
            c_mat[valor2*size + valor1] = 0;
            ++feromoniosVetor[valor1];
            ++feromoniosVetor[valor2];
        }
    }else if(leitura == 2){

    }

    int* menor = (int*) malloc(sizeof(int));
    menor[0] = size;
    int* posMenor = (int*) malloc(sizeof(int));
    posMenor[0] = 0;

    int* g_mat;
    int* g_res;
    int* g_fer;
    int* g_menor;
    int* g_posMenor;
    int* g_validacao;

    hipMalloc(&g_mat, size * size * sizeof(int));
    hipMalloc(&g_res, size * size * sizeof(int));
    hipMalloc(&g_fer, size * sizeof(int));
    hipMalloc(&g_menor, sizeof(int));
    hipMalloc(&g_posMenor, sizeof(int));
    hipMalloc(&g_validacao, size * size * sizeof(int));

    hipMemcpy(g_mat, c_mat, size * size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(g_fer, feromoniosVetor, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(g_menor, menor, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(g_posMenor, posMenor, sizeof(int), hipMemcpyHostToDevice);

    //int blockSize = 512;
    //int numBlocks = (size + blockSize - 1)/blockSize;

    // Numeros aleatorios
    hiprandState *dev_state;

    // Aloca memória na GPU
    hipMalloc((void **)&dev_state, size * sizeof(hiprandState));

    // Configura o estado inicial do gerador de números aleatórios
    setup_kernel<<<1, size>>>(dev_state, time(NULL));

    int* g_res_final;
    hipMalloc(&g_res_final, size * sizeof(int));

    int ciclos = 100;
    start = clock();
    while(ciclos > 0){
        algoritmo<<<1, size>>>(g_mat, g_res, size, g_fer, g_menor, g_posMenor, dev_state, g_validacao, g_res_final);
        atualizaFeromonios<<<1, size>>>(g_mat, g_res_final, g_posMenor, size, g_fer);
        --ciclos;
    }

    end = clock();
    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;

    int* c_res_final = (int*) malloc(size * sizeof(int));

    hipMemcpy(menor, g_menor, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(posMenor, g_posMenor, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(c_res_final, g_res_final, size * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(c_res, g_res,size * size * sizeof(int), hipMemcpyDeviceToHost);

    printf("%i\n", menor[0]);
    printf("%i\n", posMenor[0]);
    /*
    for(int i = 0; i < size; ++i){
        printf(" %i", c_res[posMenor[0]*size + i]);
    }    
    */
   for(int i = 0 ; i < size; ++i){
        if(c_res_final[i] == -1){
            break;
        }
        printf("%i ", c_res_final[i] + 1);
        //printf("\n");
   }
    printf("\nOutros resultados:\n");

    for(int i = 0; i < size; ++i){
        for(int j = 0; j < size; ++j){
            if(c_res[i*size + j] == -1){
                break;
            }
            printf("%i ", c_res[i*size + j] + 1);
        }
        printf("\n");
    }

    printf("Tempo de execução: %.8f segundos\n", cpu_time_used);

    /*
    for(int i = 0; i < size; ++i){
        for(int j = 0; j < size; ++j){
            printf("%i ", c_mat[i*size + j]);
        }
        printf("\n");
    }    
    */

   hipFree(g_fer);
   hipFree(g_mat);
   hipFree(g_menor);
   hipFree(g_posMenor);
   hipFree(g_res);
   hipFree(g_validacao);

   free(c_mat);
   free(c_res);
   free(feromoniosVetor);
   free(menor);
   free(posMenor);

    return 0;
}